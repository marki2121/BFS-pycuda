
#include <hip/hip_runtime.h>
__global__ void BFS(int* a, int* d, int* e, int* f){
    
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    d[i] = 1;

    if(a[i] != -1){
        if(d[i] != 0){
            if(f[i] == 0){
                e[i] = a[i];
            }
        }
    }
}